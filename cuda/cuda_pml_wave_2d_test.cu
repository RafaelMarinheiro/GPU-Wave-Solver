#include "hip/hip_runtime.h"
#include "cuda_pml_wave_2d.h"

#include <cstdlib>
#include <cstring>
#include <ctime>
#include <cstdio>

#include "cuda_helper.h"

#include "cuda_pml_wave_2d_kernel_test.cu"

struct Cuda_PML_Wave_2d_sim_data_t {
	Number_t xmin, ymin;
	Number_t xmax, ymax;

	Number_t dt;
	Number_t dx;
	Number_t dy;

	Number_t t;

	Number_t c;

	int nx, ny;

	Number_t pml_width;
	Number_t pml_strength;
	Number_t density;

	Number_t * ubuf;
	Number_t * ubuf_d;
	bool * isBulk;
	bool * isBulk_d;
	Number_t * gradient;
	Number_t * gradient_d;
	Number_t radius;
};

Number_t * wave_sim_get_u(Cuda_PML_Wave_2d_t wave){
	return wave->ubuf;
}

double gradient(double x, double y, int dim){
	double dist = sqrt((x-0.5)*(x-0.5) + (y-0.5)*(y-0.5));
	if(dim == 0){
		return (x-0.5)/dist;
	} else{
		return (y-0.5)/dist;
	}
}

Cuda_PML_Wave_2d_t wave_sim_init(Number_t xmin, Number_t ymin,
						Number_t xmax, Number_t ymax,
						Number_t c, Number_t dt, 
						int nx, int ny,
						Number_t (*init_function)(Number_t, Number_t, void *),
						void * ctx,
						Number_t pml_width,
						Number_t pml_strength){
	Cuda_PML_Wave_2d_t wave = (Cuda_PML_Wave_2d_t) malloc(sizeof(Cuda_PML_Wave_2d_sim_data_t));
	
	wave->xmin = xmin;
	wave->ymin = ymin;
	wave->xmax = xmax;
	wave->ymax = ymax;
	wave->radius = 0.1;
	Number_t radius = wave->radius;
	wave->c = c;

	wave->dt = dt;

	wave->t = 0;
	wave->density = 1;

	wave->nx = nx;
	wave->ny = ny;

	wave->dx = (xmax-xmin)/nx;
	wave->dy = (ymax-ymin)/ny;

	wave->pml_width = pml_width;
	wave->pml_strength = pml_strength;
	wave->ubuf = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->ubuf, 3*(wave->nx)*(wave->ny)*sizeof(Number_t)));
	wave->isBulk = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->isBulk, (wave->nx)*(wave->ny)*sizeof(bool)));
	wave->gradient = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->gradient, 2*(wave->nx)*(wave->ny)*sizeof(Number_t)));
	
	wave->ubuf_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->ubuf_d, 3*(wave->nx)*(wave->ny)*sizeof(Number_t)));
	wave->isBulk_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->isBulk_d, (wave->nx)*(wave->ny)*sizeof(bool)));
	wave->gradient_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->gradient_d, 2*(wave->nx)*(wave->ny)*sizeof(Number_t)));

	//Set the pressures
	for(int i = 0; i < ny; i++){
		Number_t y = wave_sim_get_y(wave, i);
		for(int j = 0; j < nx; j++){
			Number_t x = wave_sim_get_x(wave, j);
			if((x-0.5)*(x-0.5)+(y-0.5)*(y-0.5) < radius*radius){
				wave->isBulk[j + nx*i] = false;
			} else{
				wave->isBulk[j + nx*i] = true;
			}
			wave->gradient[2*(j + nx*i)] = gradient(x+wave->dx/2, y, 0);
			wave->gradient[2*(j + nx*i)+1] = gradient(x, y+wave->dy/2, 1);
		}
	}

	Number_t * u = wave->ubuf;
	for(int i = 0; i < ny; i++){
		Number_t y = wave_sim_get_y(wave, i);
		for(int j = 0; j < nx; j++){
			Number_t x = wave_sim_get_x(wave, j);
			u[3*(j + i*nx)] = init_function(x, y, ctx);
		}
	}
	//Set the velocities
	for(int i = 0; i < ny; i++){
		for(int j = 0; j < nx; j++){
			u[3*(j + i*nx) + 1] = 0;
			u[3*(j + i*nx) + 2] = 0;
		}
	}

	wave_sim_apply_boundary(wave);

	Number_t * u_d = wave->ubuf_d;
	cudaCheckError(hipMemcpy(u_d, u, 3*(wave->nx)*(wave->ny)*sizeof(Number_t), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(wave->isBulk_d, wave->isBulk, (wave->nx)*(wave->ny)*sizeof(bool), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(wave->gradient_d, wave->gradient, 2*(wave->nx)*(wave->ny)*sizeof(Number_t), hipMemcpyHostToDevice));

	Number_t cco[11] = {wave->c, wave->dt,
				 	wave->dx, wave->dy,
				 	wave->xmin, wave->xmax,
				 	wave->ymin, wave->ymax,
				 	wave->pml_strength,
				 	wave->pml_width,
				 	wave->density};

	hipMemcpyToSymbol(HIP_SYMBOL(kernel_constants), cco, 11*sizeof(Number_t));

	return wave;
}

void wave_sim_free(Cuda_PML_Wave_2d_t wave){
	hipHostFree(wave->ubuf);
	hipFree(wave->ubuf_d);
	free(wave);
}

Number_t wave_sim_get_x(Cuda_PML_Wave_2d_t wave, int j){
	return ((j*wave->xmax + (wave->nx - j)*wave->xmin)/wave->nx) + wave->dx/2;
}

Number_t wave_sim_get_y(Cuda_PML_Wave_2d_t wave, int i){
	return ((i*wave->ymax + (wave->ny - i)*wave->ymin)/wave->ny) + wave->dy/2;
}

void wave_sim_step(Cuda_PML_Wave_2d_t wave){
	//Copy to GPU
	Number_t * u = wave->ubuf;
	Number_t * u_d = wave->ubuf_d;

	cudaCheckError(hipMemcpy(u_d, u, 3*(wave->nx)*(wave->ny)*sizeof(Number_t), hipMemcpyHostToDevice));
	
	size_t blocks_x = ceil(wave->nx/16.0);
	size_t blocks_y = ceil(wave->ny/16.0);
	dim3 gridDim(blocks_x, blocks_y, 1);
	size_t threads_x = 16;
	size_t threads_y = 16;

	dim3 blockDim(threads_x, threads_y, 1);
	cuda_pml_wave_2d_velocity_kernel<<< gridDim, blockDim >>>(wave->ubuf_d,
															  wave->gradient_d,
															  wave->isBulk_d,
															  wave->t,
													 		  wave->nx, wave->ny);
	cudaCheckError(hipGetLastError());
	cuda_pml_wave_2d_pressure_kernel<<< gridDim, blockDim >>>(wave->ubuf_d,
															  wave->isBulk_d,
													 wave->nx, wave->ny);
	cudaCheckError(hipGetLastError());

	//Copy back
	cudaCheckError(hipMemcpy(u, u_d, 3*(wave->nx)*(wave->ny)*sizeof(Number_t), hipMemcpyDeviceToHost));
	
	wave->t += wave->dt;
	wave_sim_apply_boundary(wave);
	
}

void wave_sim_apply_boundary(Cuda_PML_Wave_2d_t wave){
	Number_t * u = wave->ubuf;
	int nx = wave->nx;
	int ny = wave->ny;

	for(int i = 0; i < ny; i++){
		u[3*(nx-1 + nx*i)+1] = 0;
	}

	for(int j = 0; j < nx; j++){
		u[3*(j + nx*(ny-1))+2] = 0;
	}
}