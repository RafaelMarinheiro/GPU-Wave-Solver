#include "hip/hip_runtime.h"
#include "cuda_PAN_wave_3d.h"

#include <cstdlib>
#include <cstring>
#include <ctime>
#include <cstdio>

#include "cuda_helper.h"

#include "cuda_PAN_wave_3d_kernel.cu"

struct Cuda_PAN_Wave_3d_sim_data_t {
	Number_t xmin, ymin, zmin;
	Number_t xmax, ymax, zmax;

	Number_t dt;

	Number_t dx, dy, dz;

	Number_t t;

	Number_t c;

	int nx, ny, nz;

	Number_t pml_width;
	Number_t pml_strength;
	Number_t density;

	Number_t * ubuf;
	Number_t * ubuf_d;

	bool * isBulk;
	bool * isBulk_d;

	Number_t * gradient;
	Number_t * gradient_d;

	int listening_count;
	Number_t * listening_positions_d;
	Number_t * listeningOutput;
	Number_t * listeningOutput_d;

	Number_t xcenter, ycenter, zcenter;
	Number_t pulse;

	bool updated;
};

Number_t * wave_sim_get_u(Cuda_PAN_Wave_3d_t wave){
	if(wave->updated){
		return wave->ubuf;
	} else{
		Number_t * u = wave->ubuf;
		Number_t * u_d = wave->ubuf_d;

		cudaCheckError(hipMemcpy(u, u_d, 4*6*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t), hipMemcpyDeviceToHost));
		wave->updated = true;
		return u;
	}
}

Cuda_PAN_Wave_3d_t wave_sim_init(Number_t xmin, Number_t ymin, Number_t zmin,
								 Number_t xmax, Number_t ymax, Number_t zmax,
								 Number_t c, Number_t dt,
								 Number_t cellsize,
								 int listening_count,
								 Number_t * listening_positions,
								 const Wave_InitialCondition3D & initial,
								 const Wave_BoundaryEvaluator3D & boundary,
								 Number_t xcenter, Number_t ycenter, Number_t zcenter,
								 const Wave_GradientEvaluator3D & gradient,
								 Number_t pml_width,
								 Number_t pml_strength,
								 Number_t pulse){
	Cuda_PAN_Wave_3d_t wave = (Cuda_PAN_Wave_3d_t) malloc(sizeof(Cuda_PAN_Wave_3d_sim_data_t));
	
	wave->xmin = xmin;
	wave->ymin = ymin;
	wave->zmin = zmin;
	wave->xmax = xmax;
	wave->ymax = ymax;
	wave->zmax = zmax;

	wave->updated = true;

	wave->xcenter = xcenter;
	wave->ycenter = ycenter;
	wave->zcenter = zcenter;

	wave->c = c;

	wave->dt = dt;

	wave->t = 0;
	wave->density = 1;

	wave->nx = ceil((xmax-xmin)/cellsize);
	wave->ny = ceil((ymax-ymin)/cellsize);
	wave->nz = ceil((zmax-zmin)/cellsize);
	int nx = wave->nx;
	int ny = wave->ny;
	int nz = wave->nz;

	wave->dx = (xmax-xmin)/wave->nx;
	wave->dy = (ymax-ymin)/wave->ny;
	wave->dz = (zmax-zmin)/wave->nz;

	wave->pml_width = pml_width;
	wave->pml_strength = pml_strength;
	wave->pulse = pulse;

	wave->ubuf = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->ubuf, 4*6*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t)));
	wave->ubuf_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->ubuf_d, 4*6*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t)));

	wave->isBulk = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->isBulk, (wave->nx)*(wave->ny)*(wave->nz)*sizeof(bool)));
	wave->isBulk_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->isBulk_d, (wave->nx)*(wave->ny)*(wave->nz)*sizeof(bool)));

	wave->gradient = NULL;
	cudaCheckError(hipHostMalloc((void**)&wave->gradient, 3*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t)));
	wave->gradient_d = NULL;
	cudaCheckError(hipMalloc((void**)&wave->gradient_d, 3*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t)));

	if(listening_count > 0){
		wave->listening_count = listening_count;

		wave->listening_positions_d = NULL;
		cudaCheckError(hipMalloc((void**)&wave->listening_positions_d, 3*listening_count*sizeof(Number_t)));

		wave->listeningOutput = NULL;
		cudaCheckError(hipHostMalloc((void**)&wave->listeningOutput, 6*listening_count*sizeof(Number_t)));
		wave->listeningOutput_d = NULL;
		cudaCheckError(hipMalloc((void**)&wave->listeningOutput_d, 6*listening_count*sizeof(Number_t)));
	}

	for(int k = 0; k < nz; k++){
		Number_t z = wave_sim_get_z(wave, k);
		for(int j = 0; j < ny; j++){
			Number_t y = wave_sim_get_y(wave, j);
			for(int i = 0; i < nx; i++){
				Number_t x = wave_sim_get_x(wave, i);
				if(boundary(x, y, z)){
					wave->isBulk[(i + nx*(j + ny*k))] = false;
				} else{
					wave->isBulk[(i + nx*(j + ny*k))] = true;
				}
				int idx = 3*(i + nx*(j + ny*k));
				wave->gradient[idx] = gradient(x+wave->dx/2, y, z, 0);
				wave->gradient[idx+1] = gradient(x, y+wave->dy/2, z, 1);
				wave->gradient[idx+2] = gradient(x, y, z+wave->dz/2, 2);
			}
		}
	}

	//Set the pressures
	Number_t * u = wave->ubuf;

	memset(u, 0, 4*6*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t));

	for(int k = 0; k < nz; k++){
		Number_t z = wave_sim_get_z(wave, k);
		for(int j = 0; j < ny; j++){
			Number_t y = wave_sim_get_y(wave, j);
			for(int i = 0; i < nx; i++){
				Number_t x = wave_sim_get_x(wave, i);
				Number_t val = initial(x, y, z);
				int idx = 4*(i + nx*(j + ny*k));
				int stride = 4*nx*ny*nz;
				u[idx] = val;
				u[idx+stride] = val;
				u[idx+2*stride] = val;
				u[idx+3*stride] = val;
				u[idx+4*stride] = val;
				u[idx+5*stride] = val;
			}
		}
	}

	cudaCheckError(hipMemcpy(wave->ubuf_d, wave->ubuf, 4*6*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(wave->isBulk_d, wave->isBulk, (wave->nx)*(wave->ny)*(wave->nz)*sizeof(bool), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(wave->gradient_d, wave->gradient, 3*(wave->nx)*(wave->ny)*(wave->nz)*sizeof(Number_t), hipMemcpyHostToDevice));
	if(listening_count > 0){
		cudaCheckError(hipMemcpy(wave->listening_positions_d, listening_positions, 3*listening_count*sizeof(Number_t), hipMemcpyHostToDevice));	
	}
	
	Number_t cco[18] = {wave->c, wave->dt,
					 	wave->dx, wave->dy,
					 	wave->xmin, wave->xmax,
					 	wave->ymin, wave->ymax,
					 	wave->zmin, wave->zmax,
					 	wave->pml_strength,
					 	wave->pml_width,
					 	wave->density,
					 	wave->xcenter,
					 	wave->ycenter,
					 	wave->zcenter,
					 	wave->pulse,
					 	wave->dz};

	hipMemcpyToSymbol(HIP_SYMBOL(kernel_constants), cco, 18*sizeof(Number_t));

	return wave;
}

void wave_sim_free(Cuda_PAN_Wave_3d_t wave){
	hipHostFree(wave->ubuf);
	hipHostFree(wave->isBulk);
	hipHostFree(wave->gradient);
	hipHostFree(wave->listeningOutput);
	hipFree(wave->ubuf_d);
	hipFree(wave->isBulk_d);
	hipFree(wave->gradient_d);
	hipFree(wave->listeningOutput_d);
	hipFree(wave->listening_positions_d);
	free(wave);
}

Number_t wave_sim_get_x(Cuda_PAN_Wave_3d_t wave, int i){
	return ((i*wave->xmax + (wave->nx - i)*wave->xmin)/wave->nx) + wave->dx/2;
}

Number_t wave_sim_get_y(Cuda_PAN_Wave_3d_t wave, int j){
	return ((j*wave->ymax + (wave->ny - j)*wave->ymin)/wave->ny) + wave->dy/2;
}

Number_t wave_sim_get_z(Cuda_PAN_Wave_3d_t wave, int k){
	return ((k*wave->zmax + (wave->nz - k)*wave->zmin)/wave->nz) + wave->dz/2;
}


void wave_sim_step(Cuda_PAN_Wave_3d_t wave){
	size_t blocks_x = ceil(wave->nx/16.0);
	size_t blocks_y = ceil(wave->ny/16.0);
	dim3 gridDim(blocks_x, blocks_y, 1);
	size_t threads_x = 16;
	size_t threads_y = 16;

	dim3 blockDim(threads_x, threads_y, 1);
	int stride = 4*(wave->nx)*(wave->ny)*(wave->nz);
	for(int i = 0; i < 6; i++){
		cuda_pan_wave_3d_velocity_kernel<<< gridDim, blockDim >>>(wave->ubuf_d+i*stride,
																  wave->gradient_d,
																  wave->isBulk_d,
																  wave->t,
														 		  wave->nx,
														 		  wave->ny,
														 		  wave->nz,
														 		  i);
		cudaCheckError(hipGetLastError());
		cuda_pan_wave_3d_pressure_kernel<<< gridDim, blockDim >>>(wave->ubuf_d+i*stride,
																  wave->isBulk_d,
																  wave->nx,
																  wave->ny,
																  wave->nz);
	}
	cudaCheckError(hipGetLastError());

	wave->updated = false;
	wave->t += wave->dt;
}

Number_t * wave_listen(Cuda_PAN_Wave_3d_t wave, int field){
	size_t blocks_x = ceil(wave->listening_count/256.0);
	dim3 gridDim(blocks_x, 1, 1);
	size_t threads_x = 256;

	dim3 blockDim(threads_x, 1, 1);
	int stride = 4*(wave->nx)*(wave->ny)*(wave->nz)*field;
	int linstride = wave->listening_count*field;

	cuda_pan_wave_3d_listen_kernel<<<gridDim, blockDim>>>(wave->ubuf_d+stride,
														  wave->listeningOutput_d,
														  wave->listening_count,
														  wave->listening_positions_d,
														  wave->nx,
														  wave->ny,
														  wave->nz);
	cudaCheckError(hipGetLastError());

	//Copy back
	cudaCheckError(hipMemcpy(wave->listeningOutput+linstride, wave->listeningOutput_d, wave->listening_count*sizeof(Number_t), hipMemcpyDeviceToHost));
	return wave->listeningOutput+linstride;
}

void wave_sim_get_divisions(const Cuda_PAN_Wave_3d_t wave, int * nx, int * ny, int * nz){
	(*nx) = wave->nx;
	(*ny) = wave->ny;
	(*nz) = wave->nz;
}

Number_t wave_sim_get_current_time(const Cuda_PAN_Wave_3d_t wave){
	return wave->t;
}

void wave_sim_get_bounds(const Cuda_PAN_Wave_3d_t wave,
						 Number_t * xmin, Number_t * xmax,
						 Number_t * ymin, Number_t * ymax,
						 Number_t * zmin, Number_t * zmax){
	(*xmin) = wave->xmin;
	(*xmax) = wave->xmax;
	(*ymin) = wave->ymin;
	(*ymax) = wave->ymax;
	(*zmin) = wave->zmin;
	(*zmax) = wave->zmax;
}