#include "hip/hip_runtime.h"
__constant__ Number_t kernel_constants[18];

#include "cuda_PAN_wave_3d_kernel_math.cu"

__device__ __forceinline__ Number_t w_get_pos(int j, int nn, Number_t vmin, Number_t vmax, Number_t dd){
	return ((j*vmax + (nn - j)*vmin)/nn) + dd/2;
}

__global__ void cuda_pan_wave_3d_velocity_kernel(Number_t * __restrict__ u,
												 const Number_t * __restrict__ grad,
												 const bool * __restrict__ isBulk,
												 Number_t t,
												 const int nx,
												 const int ny,
												 const int nz){
	const Number_t dt = kernel_constants[1];
	const Number_t idt = 1/dt;
	const Number_t dx = kernel_constants[2];
	const Number_t dy = kernel_constants[3];
	const Number_t xmin = kernel_constants[4];
	const Number_t xmax = kernel_constants[5];
	const Number_t ymin = kernel_constants[6];
	const Number_t ymax = kernel_constants[7];
	const Number_t zmin = kernel_constants[8];
	const Number_t zmax = kernel_constants[9];
	const Number_t pml_strength = kernel_constants[10];
	const Number_t pml_width = kernel_constants[11];
	const Number_t density = kernel_constants[12];
	const Number_t timepulse = kernel_constants[16];
	const Number_t dz = kernel_constants[17];
	const Number_t mit = PAN_Mitchelli(t, timepulse);


	Number_t local_z[4];
	Number_t local_old[4];
	Number_t local_new[4];

	bool ibulk;
	bool bulk_z;

	__shared__ Number_t cache[18][18];
	__shared__ bool cache_bulk[18][18];

	const int bdx = 16;
	const int bdy = 16;

	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i < nx && j < ny){
		const Number_t bx = w_get_pos(i, nx, xmin, xmax, dx);
		const Number_t by = w_get_pos(j, ny, ymin, ymax, dy);

		const int tx = threadIdx.x + 1;
		const int ty = threadIdx.y + 1;

		//Compute the first guy:
		const int bbbase = 4*(i + nx*j);

		local_z[0] = u[bbbase+0];
		local_z[1] = u[bbbase+1];
		local_z[2] = u[bbbase+2];
		local_z[3] = u[bbbase+3];
		bulk_z = isBulk[(i + nx*j)];

		#pragma unroll
		for(int other = 0; other < nz; other++){
			int k = other;
			int base = i + nx*(j + ny*k);
			int idx = 4*base;
			const Number_t bz = w_get_pos(k, nz, zmin, zmax, dz);

			local_old[0] = local_z[0];
			local_old[1] = local_z[1];
			local_old[2] = local_z[2];
			local_old[3] = local_z[3];
			local_new[0] = local_old[0];
			local_new[1] = local_new[2] = local_new[3] = 0;
			ibulk = bulk_z;

			cache[tx][ty] = local_old[0];
			cache_bulk[tx][ty] = ibulk;

			if(threadIdx.x == 0){
				if(i+bdx < nx){
					const int base = 4*((i+bdx) + nx*(j + ny*k));
					cache[tx+bdx][ty] = u[base+0];
					cache_bulk[tx+bdx][ty] = isBulk[(i+bdx) + nx*(j + ny*k)];
				} else{
					cache[tx+bdx][ty] = 0;
					cache_bulk[tx+bdx][ty] = false;
				}
			}
			if(threadIdx.y == 0){
				if(j+bdy < ny){
					const int base = 4*(i + nx*((j+bdy) + ny*k));
					cache[tx][ty+bdy] = u[base+0];
					cache_bulk[tx][ty+bdy] = isBulk[(i + nx*((j+bdy) + ny*k))];
				} else{
					cache[tx][ty+bdy] = 0;
					cache_bulk[tx][ty+bdy] = false;
				}
			}
			__syncthreads();

			//Solve for X
			if(i != nx-1){
				const bool otherbulk = cache_bulk[tx+1][ty];
				if(ibulk && otherbulk){
					const Number_t absortion = pan_wave_3d_absortion(bx+dx/2, xmin, xmax, pml_strength, pml_width);
					const Number_t update = pan_wave_3d_vel_update(idt, absortion);
					const Number_t gradient = pan_wave_3d_gradient(idt, absortion, dx, density);
					local_new[1] = local_old[1]*update + gradient*(cache[tx+1][ty] - cache[tx][ty]);
				} else if(ibulk || otherbulk){
					Number_t gradi = grad[3*base]*mit;
					if(ibulk){
						gradi = -gradi;
					}
					local_new[1] = local_old[1] + gradi*PAN_boundary(bx+dx/2, by, bz, 0, 0);
				}
			}
			//Solve for Y
			if(j != ny-1){
				const bool otherbulk = cache_bulk[tx][ty+1];
				if(ibulk && otherbulk){
					const Number_t absortion = pan_wave_3d_absortion(by+dy/2, ymin, ymax, pml_strength, pml_width);
					const Number_t update = pan_wave_3d_vel_update(idt, absortion);
					const Number_t gradient = pan_wave_3d_gradient(idt, absortion, dy, density);
					local_new[2] = local_old[2]*update + gradient*(cache[tx][ty+1] - cache[tx][ty]);
				} else if(ibulk || otherbulk){
					Number_t gradi = grad[3*base+1]*mit;
					if(ibulk){
						gradi = -gradi;
					}
					local_new[2] = local_old[2] + gradi*PAN_boundary(bx, by+dy/2, bz, 0, 1);
				}
			}

			//Solve for Z
			if(k != nz-1){
				const int bbase = (i + nx*(j + ny*(k+1)));
				const int bidx = 4*bbase;
				local_z[0] = u[bidx+0];
				local_z[1] = u[bidx+1];
				local_z[2] = u[bidx+2];
				local_z[3] = u[bidx+3];
				bulk_z = isBulk[bbase];
				const bool otherbulk = bulk_z;
				if(ibulk && otherbulk){
					const Number_t absortion = pan_wave_3d_absortion(bz+dz/2, zmin, zmax, pml_strength, pml_width);
					const Number_t update = pan_wave_3d_vel_update(idt, absortion);
					const Number_t gradient = pan_wave_3d_gradient(idt, absortion, dz, density);
					local_new[3] = local_old[3]*update + gradient*(local_z[0] - cache[tx][ty]);
				} else if(ibulk || otherbulk){
					Number_t gradi = grad[3*base+2]*mit;
					if(ibulk){
						gradi = -gradi;
					}
					local_new[3] = local_old[3] + gradi*PAN_boundary(bx, by, bz+dz/2, 0, 2);
				}
			}
			//u[idx + 0] = local_new[0];
			u[idx + 1] = local_new[1];
			u[idx + 2] = local_new[2];
			u[idx + 3] = local_new[3];
		}
	}
}


__global__ void cuda_pan_wave_3d_pressure_kernel(Number_t * u,
												 bool * isBulk,
												 const int nx,
												 const int ny,
												 const int nz){

	Number_t local_me[6];
	Number_t local_new[6];
	Number_t local_v_me[6];
	Number_t local_v_other[6];

	Number_t c = kernel_constants[0];
	Number_t dt = kernel_constants[1];
	Number_t idt = 1/dt;
	Number_t dx = kernel_constants[2];
	Number_t dy = kernel_constants[3];
	Number_t xmin = kernel_constants[4];
	Number_t xmax = kernel_constants[5];
	Number_t ymin = kernel_constants[6];
	Number_t ymax = kernel_constants[7];
	Number_t zmin = kernel_constants[8];
	Number_t zmax = kernel_constants[9];
	Number_t pml_strength = kernel_constants[10];
	Number_t pml_width = kernel_constants[11];
	Number_t density = kernel_constants[12];
	Number_t dz = kernel_constants[17];

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i < nx && j < ny){
		for(int other = 0; other < nz; other++){
			int k = other;
			int base = i + nx*(j+ny*k);
			if(isBulk[base]){
				int idx = 4*6*base;
				Number_t update = 0;
				Number_t local_div[6] = {0, 0, 0, 0, 0, 0};
				Number_t abs_d;
				Number_t dir_d;
				Number_t upd_d;
				Number_t div_d;
				local_me[0] = u[idx+0];
				local_me[1] = u[idx+1];
				local_me[2] = u[idx+2];
				local_me[3] = u[idx+3];
				local_me[4] = u[idx+4];
				local_me[5] = u[idx+5];

				//Solve for X
				{
					local_v_me[0] = u[idx+6];
					local_v_me[1] = u[idx+7];
					local_v_me[2] = u[idx+8];
					local_v_me[3] = u[idx+9];
					local_v_me[4] = u[idx+10];
					local_v_me[5] = u[idx+11];
					if(i != 0){
						int bbase = (i-1) + nx*(j+ny*k);
						int bidx = 4*6*bbase + 6;
						local_v_other[0] = u[bidx + 0];
						local_v_other[1] = u[bidx + 1];
						local_v_other[2] = u[bidx + 2];
						local_v_other[3] = u[bidx + 3];
						local_v_other[4] = u[bidx + 4];
						local_v_other[5] = u[bidx + 5];
					} else{
						local_v_other[0] = 0;
						local_v_other[1] = 0;
						local_v_other[2] = 0;
						local_v_other[3] = 0;
						local_v_other[4] = 0;
						local_v_other[5] = 0;
					}

					Number_t bx = w_get_pos(i, nx, xmin, xmax, dx);
					abs_d = pan_wave_3d_absortion(bx+dx/2, xmin, xmax, pml_strength, pml_width);
					dir_d = pan_wave_3d_directional(idt, abs_d);
					upd_d = pan_wave_3d_pre_update(idt, abs_d, dir_d);
					div_d = pan_wave_3d_pre_divergence(density, c, dir_d, dx);

					update += upd_d/3;
					local_div[0] += div_d*(local_v_me[0] - local_v_other[0]);
					local_div[1] += div_d*(local_v_me[1] - local_v_other[1]);
					local_div[2] += div_d*(local_v_me[2] - local_v_other[2]);
					local_div[3] += div_d*(local_v_me[3] - local_v_other[3]);
					local_div[4] += div_d*(local_v_me[4] - local_v_other[4]);
					local_div[5] += div_d*(local_v_me[5] - local_v_other[5]);
				}

				//Solve for Y
				{
					local_v_me[0] = u[idx+12];
					local_v_me[1] = u[idx+13];
					local_v_me[2] = u[idx+14];
					local_v_me[3] = u[idx+15];
					local_v_me[4] = u[idx+16];
					local_v_me[5] = u[idx+17];
					if(j != 0){
						int bbase = i + nx*((j-1)+ny*k);
						int bidx = 4*6*bbase + 12;
						local_v_other[0] = u[bidx + 0];
						local_v_other[1] = u[bidx + 1];
						local_v_other[2] = u[bidx + 2];
						local_v_other[3] = u[bidx + 3];
						local_v_other[4] = u[bidx + 4];
						local_v_other[5] = u[bidx + 5];
					} else{
						local_v_other[0] = 0;
						local_v_other[1] = 0;
						local_v_other[2] = 0;
						local_v_other[3] = 0;
						local_v_other[4] = 0;
						local_v_other[5] = 0;
					}

					Number_t by = w_get_pos(j, ny, ymin, ymax, dy);
					abs_d = pan_wave_3d_absortion(by+dy/2, ymin, ymax, pml_strength, pml_width);
					dir_d = pan_wave_3d_directional(idt, abs_d);
					upd_d = pan_wave_3d_pre_update(idt, abs_d, dir_d);
					div_d = pan_wave_3d_pre_divergence(density, c, dir_d, dy);

					update += upd_d/3;
					local_div[0] += div_d*(local_v_me[0] - local_v_other[0]);
					local_div[1] += div_d*(local_v_me[1] - local_v_other[1]);
					local_div[2] += div_d*(local_v_me[2] - local_v_other[2]);
					local_div[3] += div_d*(local_v_me[3] - local_v_other[3]);
					local_div[4] += div_d*(local_v_me[4] - local_v_other[4]);
					local_div[5] += div_d*(local_v_me[5] - local_v_other[5]);
				}

				//Solve for Z
				{
					local_v_me[0] = u[idx+18];
					local_v_me[1] = u[idx+19];
					local_v_me[2] = u[idx+20];
					local_v_me[3] = u[idx+21];
					local_v_me[4] = u[idx+22];
					local_v_me[5] = u[idx+23];
					if(k != 0){
						int bbase = i + nx*(j+ny*(k-1));
						int bidx = 4*6*bbase + 18;
						local_v_other[0] = u[bidx + 0];
						local_v_other[1] = u[bidx + 1];
						local_v_other[2] = u[bidx + 2];
						local_v_other[3] = u[bidx + 3];
						local_v_other[4] = u[bidx + 4];
						local_v_other[5] = u[bidx + 5];
					} else{
						local_v_other[0] = 0;
						local_v_other[1] = 0;
						local_v_other[2] = 0;
						local_v_other[3] = 0;
						local_v_other[4] = 0;
						local_v_other[5] = 0;
					}

					Number_t bz = w_get_pos(k, nz, zmin, zmax, dz);
					abs_d = pan_wave_3d_absortion(bz+dz/2, zmin, zmax, pml_strength, pml_width);
					dir_d = pan_wave_3d_directional(idt, abs_d);
					upd_d = pan_wave_3d_pre_update(idt, abs_d, dir_d);
					div_d = pan_wave_3d_pre_divergence(density, c, dir_d, dz);

					update += upd_d/3;
					local_div[0] += div_d*(local_v_me[0] - local_v_other[0]);
					local_div[1] += div_d*(local_v_me[1] - local_v_other[1]);
					local_div[2] += div_d*(local_v_me[2] - local_v_other[2]);
					local_div[3] += div_d*(local_v_me[3] - local_v_other[3]);
					local_div[4] += div_d*(local_v_me[4] - local_v_other[4]);
					local_div[5] += div_d*(local_v_me[5] - local_v_other[5]);
				}
				local_new[0] = update*local_me[0] + local_div[0];
				local_new[1] = update*local_me[1] + local_div[1];
				local_new[2] = update*local_me[2] + local_div[2];
				local_new[3] = update*local_me[3] + local_div[3];
				local_new[4] = update*local_me[4] + local_div[4];
				local_new[5] = update*local_me[5] + local_div[5];

				u[idx+0] = local_new[0];
				u[idx+1] = local_new[1];
				u[idx+2] = local_new[2];
				u[idx+3] = local_new[3];	
				u[idx+4] = local_new[4];
				u[idx+5] = local_new[5];
			}
		}
	}
}